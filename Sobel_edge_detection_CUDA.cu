#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <fcntl.h>
#include <hip/hip_runtime.h>
#include "string.h"

#define DEFAULT_THRESHOLD 4000

#define DEFAULT_FILENAME "ansel3.ppm"

__global__ void sobel(unsigned int *ingoing, int *outgoing, int xsize, int ysize, int threshold) {

	 int x = threadIdx.x + (blockIdx.x * blockDim.x);
	 int y = threadIdx.y + (blockIdx.y * blockDim.y);

	 if ((x > 0) && (x < ysize - 1) &&(y > 0) &&  (y < xsize - 1)) {

		  int sum_x = ingoing[(x + 1) + ((y - 1) * ysize)]  -		ingoing[(x - 1) + ((y - 1) * ysize)]
				 + (2 * ingoing[(x + 1) + (		y * ysize)]) - (2 * ingoing[(x - 1) + (		y * ysize)])
						+ ingoing[(x + 1) + ((y + 1) * ysize)]  -		ingoing[(x - 1) + ((y + 1) * ysize)];

		  int sum_y = ingoing[(x + 1) + ((y + 1) * ysize)]  -		ingoing[(x + 1) + ((y - 1) * ysize)]
				 + (2 * ingoing[	  x  + ((y + 1) * ysize)]) - (2 * ingoing[		x + ((y - 1) * ysize)])
						+ ingoing[(x - 1) + ((y + 1) * ysize)]  -		ingoing[(x - 1) + ((y - 1) * ysize)];

		  
		  int magnitude = (sum_x * sum_x) + (sum_y * sum_y);
		  int i = x + (y * ysize);
		  if (magnitude > threshold) 
				outgoing[i] = 255;
		  

	 }
}

unsigned int *read_ppm(char *filename, int *xsize, int *ysize, int *maxval) {

	 if (!filename || filename[0] == '\0') {
		  fprintf(stderr, "read_ppm but no file name\n");
		  return NULL;
	 }

	 FILE *fp;

	 fprintf(stderr, "read_ppm(%s)\n", filename);
	 fp = fopen(filename, "rb");
	 if (!fp) {
		  fprintf(stderr, "read_ppm() ERROR file '%s' cannot be opened for reading\n", filename);
		  return NULL;
	 }

	 char chars[1024];
	 int num = fread(chars, sizeof(char), 1000, fp);

	 if (chars[0] != 'P' || chars[1] != '6') {
		  fprintf(stderr, "Texture::Texture() ERROR file '%s' does not start with \"P6\" I am expecting a binary PPM file\n", filename);
		  return NULL;
	 }

	 unsigned int width, height, maxvalue;

	 char *ptr = chars + 3; // P 6 newline
	 if (*ptr == '#') { // comment line!
		  ptr = 1 + strstr(ptr, "\n");
	 }

	 num = sscanf(ptr, "%d\n%d\n%d",  &width, &height, &maxvalue);
	 fprintf(stderr, "read %d things: width %d, height %d, maxval %d\n", num, width, height, maxvalue);
	 *xsize = width;
	 *ysize = height;
	 *maxval = maxvalue;

	 unsigned int *pic = (unsigned int *)malloc(width * height * sizeof(unsigned int));
	 if (!pic) {
		  fprintf(stderr, "read_ppm()  unable to allocate %d x %d unsigned ints for the picture\n", width, height);
		  return NULL; // fail but return
	 }

	 // allocate buffer to read the rest of the file into
	 int bufsize =  3 * width * height * sizeof(unsigned char);
	 if ((*maxval) > 255) {
		  bufsize *= 2;
	 }

	 unsigned char *buf = (unsigned char *)malloc(bufsize);
	 if (!buf) {
		  fprintf(stderr, "read_ppm()  unable to allocate %d bytes of read buffer\n", bufsize);
		  return NULL; // fail but return
	 }

	 // TODO really read
	 char duh[80];
	 char *line = chars;

	 // find the start of the pixel data.	no doubt stupid
	 sprintf(duh, "%d\0", *xsize);
	 line = strstr(line, duh);
	 //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
	 line += strlen(duh) + 1;

	 sprintf(duh, "%d\0", *ysize);
	 line = strstr(line, duh);
	 //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
	 line += strlen(duh) + 1;

	 sprintf(duh, "%d\0", *maxval);
	 line = strstr(line, duh);

	 fprintf(stderr, "%s found at offset %d\n", duh, line - chars);
	 line += strlen(duh) + 1;

	 long offset = line - chars;
	 //lseek(fd, offset, SEEK_SET); // move to the correct offset
	 fseek(fp, offset, SEEK_SET); // move to the correct offset
	 //long numread = read(fd, buf, bufsize);
	 long numread = fread(buf, sizeof(char), bufsize, fp);
	 fprintf(stderr, "Texture %s	read %ld of %ld bytes\n", filename, numread, bufsize);

	 fclose(fp);

	 int pixels = (*xsize) * (*ysize);
	 int i;
	 for (i=0; i<pixels; i++) {
		  pic[i] = (int) buf[3*i];  // red channel
	 }

	 return pic; // success

}

void write_ppm( char *filename, int xsize, int ysize, int maxval, int *pic) {

	 FILE *fp;

	 fp = fopen(filename, "w");
	 if (!fp) {
		  fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n");
		  exit(-1);
	 }

	 fprintf(fp, "P6\n");
	 fprintf(fp,"%d %d\n%d\n", xsize, ysize, maxval);

	 int numpix = xsize * ysize;
	 int i;
	 for (i=0; i<numpix; i++) {
		  unsigned char uc = (unsigned char) pic[i];
		  fprintf(fp, "%c%c%c", uc, uc, uc);
	 }

	 fclose(fp);

}

int main(int argc, char **argv) {

	 char *filename;
	 filename = strdup(DEFAULT_FILENAME);

	 int threshold;
	 threshold = DEFAULT_THRESHOLD;

	 if (argc > 1) {
		  if (argc == 3) {
				filename = strdup(argv[1]);
				threshold = atoi(argv[2]);
		  }
		  if (argc == 2) {
				threshold = atoi(argv[1]);
		  }
	 }

	 int xsize, ysize, maxval;
	 unsigned int *pic = read_ppm(filename, &ysize, &xsize, &maxval);

	 int size = xsize * ysize;

	 dim3 BLOCK(32, 32);

	 dim3 GRID((int)ceil((float)ysize / 32), (int)ceil((float)xsize / 32));

	 unsigned int *h_ingoing;

	 int *h_outgoing;

	 h_ingoing = pic;

	 h_outgoing = (int *)calloc(size, sizeof *h_outgoing);

	 unsigned int *d_ingoing;

	 int *d_outgoing;

	 hipMalloc(&d_ingoing, size * sizeof *d_ingoing);

	 hipMalloc(&d_outgoing, size * sizeof *d_outgoing);

	 hipMemcpy(d_ingoing, h_ingoing, size * sizeof *h_ingoing, hipMemcpyHostToDevice);

	 hipMemcpy(d_outgoing, h_outgoing, size * sizeof *h_outgoing, hipMemcpyHostToDevice);

	 float time;

	 hipEvent_t begin, end;

	 hipEventCreate(&begin);

	 hipEventCreate(&end);
	 hipEventRecord(begin, 0);

	 sobel<<<GRID, BLOCK>>>(d_ingoing, d_outgoing, xsize, ysize, threshold);

	 hipEventRecord(end, 0);


	 hipEventSynchronize(end);


	 hipEventElapsedTime(&time, begin, end);

	 hipMemcpy(h_outgoing, d_outgoing, size * sizeof *h_outgoing, hipMemcpyDeviceToHost);

	 printf("%f\n", time);

	 write_ppm("result.ppm", ysize, xsize, 255, h_outgoing);
	 
}

